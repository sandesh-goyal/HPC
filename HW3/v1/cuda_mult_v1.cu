//v1.0
//**NO MEMORY OPTIMIZATION, NO SHARED MEMORY**

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100								// Square Matrix dimension
#define BLOCK_DIM 25 						// max 32 for 1024 threads in a block, 25 is a good divisor for N
#define L 0									// LOW range for Random number
#define H 65535								// HIGH range for Random number
#define RAND_MAX 2147483647					// 0x7fffffff

//-----------------------------------------------------------------Function Declration
void random_ints(float a[N*N], float b[N*N]);
void print_mat(float arr[N*N]);
//-----------------------------------------------------------------GPU kernel for Multiplication
__global__ void mult(float *a, float *b, float *c)
{
	int rowid = (blockDim.y * blockIdx.y) + threadIdx.y;
	int columnid = (blockDim.x * blockIdx.x) + threadIdx.x;

	float sum = 0.0;
	for(int i = 0; i < N; i++)
	{
		sum += a[(rowid*N) + i] * b[(i*N) + columnid];
	}
	c[(rowid*N) + columnid] = sum;
}
//-----------------------------------------------------------------Main function
int main()
{	
	// Device copies of a, b, c
	float *dev_a, *dev_b, *dev_c; 	
	
	// 1D array size equivalent for 2D matrix		
	int size = N * N * sizeof(float); 
							
	// Allocate device copies of a, b, c			
	hipMalloc((void**)&dev_a, size);		
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	
	// Host copies of a, b, c, declared as 1D array instead of 2D
	// to avoid issue with pointer/address with cudaMemcpy
	float *a = (float *)malloc(size);		
	float *b = (float *)malloc(size);		
	float *c = (float *)malloc(size);
	
	// Initialize array a and b
	random_ints(a,b);						
	
	// Copy a and b to device
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	
	// 2D allocation of threads in block and blocks in grid, third dimesion is 1
	dim3 gridsize (N/BLOCK_DIM, N/BLOCK_DIM, 1);
  	dim3 blocksize(BLOCK_DIM, BLOCK_DIM, 1);
	
	// Launch mult() kernel with multiple blocks and threads
	mult<<<gridsize,blocksize >>>(dev_a, dev_b, dev_c);
	
	// Copy device result back to host copy of c
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	
	// Print a, b, and result c=ab
	//print_mat(a);
	//print_mat(b);
	//print_mat(c);
	
	// Free host memory for a, b and c
	free(a);
	free(b);
	free(c);
	
	// Free device memory for a, b and c
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
//-----------------------------------------------------------------Function definition for Random initialization
void random_ints(float a[N*N], float b[N*N])
{	
	int i;
	
    for(i=0; i<N*N; i++)
	{
		a[i] = (H-L)*((float) rand()/RAND_MAX);
		b[i] = (H-L)*((float) rand()/RAND_MAX);
	}
}
//-----------------------------------------------------------------Function definition for Print
void print_mat(float arr[N*N])
{	
	int i;
	for(i=0; i<N*N; i++)
	{
		printf("%f ", arr[i]);
		
		if((i+1)%N == 0)
			printf("\n");
	}
	printf("\n");
}
