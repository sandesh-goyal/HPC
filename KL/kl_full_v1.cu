/*
v1
for calculating maximum gain, square root method is used to get number of parallel threads calculating maximum gain
better for less number of MODULES
*/

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define MAX_TH 1024
//GPU KERNEL-----------------------------------------------------------------------CALCULATE INITIAL GAIN
__global__ void calc_init_gain(int *set1, int *set2, int *set1_int_gain, int *set1_ext_gain, int *set2_int_gain, int *set2_ext_gain, int *set1_d_gain, int *set2_d_gain, int *set1_id, int *set2_id, int set1_size, int set2_size, int MODULES)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j;
	
	if(i < set1_size)
	{
		set1_ext_gain[i] = 0;
    	set1_int_gain[i] = 0;
    	set2_ext_gain[i] = 0;
    	set2_int_gain[i] = 0;
		for(j=0; j<set1_size; j++)
		{
			set1_int_gain[i] += set1[(j*MODULES) + set1_id[i]];
			if (i< set2_size)
			{
				set2_ext_gain[i] += set1[(j*MODULES) + set2_id[i]];
				if (j < set2_size)
					set2_int_gain[i] += set2[(j*MODULES) + set2_id[i]];
			}
			if (j < set2_size)
			{
				set1_ext_gain[i] += set2[(j*MODULES) + set1_id[i]];
				//*initial_cutset_size += set2[(j*MODULES) + set1_id[i]];
			}
		}
		set1_d_gain[i] = set1_ext_gain[i] - set1_int_gain[i];
		if (i < set2_size)
			set2_d_gain[i] = set2_ext_gain[i] - set2_int_gain[i];
    }
}
//GPU KERNEL-----------------------------------------------------------------------CALCULATE GAIN BENEFIT
__global__ void cal_gain_benefit(int *set1, int *set2_id, int *set1_d_gain, int *set2_d_gain, int *gain_benefit, int set1_size, int set2_size, int k, int MODULES)
{
	int i = (blockDim.y * blockIdx.y) + threadIdx.y;
	int j = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	if((i<set1_size-k) && (j<set2_size-k))
	{
		gain_benefit[((i*(set2_size-k))+j)*3 + 0] = i;
		gain_benefit[((i*(set2_size-k))+j)*3 + 1] = j;
		gain_benefit[((i*(set2_size-k))+j)*3 + 2] = set1_d_gain[i] + set2_d_gain[j] - (2*set1[(i*MODULES) + set2_id[j]]);
	}
}
//GPU KERNEL-----------------------------------------------------------------------CALCULATE MAX GAIN TEMP
__global__ void cal_max_g_t(int *gain_benefit, int *max_g, int *max_g_location, int t, int temp, int thread_num)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < thread_num)
	{
		int i, loc;
		loc = (tid+1)*temp;
		int my_max = gain_benefit[(tid*temp*3) + 2];
		int my_max_location = tid*temp;
		
		for(i=tid*temp; i<loc; i++)
		{
			if(i < t)
			{
				if(gain_benefit[i*3 + 2] > my_max)
				{
					my_max = gain_benefit[i*3 + 2];
					my_max_location = i;
				}
			}
		}
	
		max_g[tid] = my_max;
		max_g_location[tid] = my_max_location;
	}
}
//GPU KERNEL-----------------------------------------------------------------------CALCULATE MAX GAIN
__global__ void cal_max_g(int *max_g, int *max_g_location, int *maxt, int *max_location, int thread_num)
{
	int i;
	
	*maxt = max_g[0];
	*max_location = max_g_location[0];

	for(i=1; i<thread_num; i++)
	{
		if(max_g[i] > *maxt)
		{
			*maxt = max_g[i];
			*max_location = max_g_location[i];
		}
	}
}
//GPU KERNEL-----------------------------------------------------------------------UPDATE ITERATION
__global__ void update_iteration(int *iteration, int *set1_id, int *set2_id, int *gain_benefit, int *max_location, int *maxt, int k)
{
	iteration[(k*3) + 0] = set1_id[gain_benefit[(*max_location)*3 + 0]];
    iteration[(k*3) + 1] = set2_id[gain_benefit[(*max_location)*3 + 1]];
    iteration[(k*3) + 2] = *maxt;
    /*
    printf("----------------------Iteration %d\n", k+1);
	printf("max gain benefit: %d\n", iteration[(k*3) + 2]);
    printf("set1 swapped id: %d\n", iteration[(k*3) + 0]);
    printf("set2 swapped id: %d\n", iteration[(k*3) + 1]);
    */
}
//GPU KERNEL-----------------------------------------------------------------------GAIN SWAP
__global__ void gain_swap(int *set1, int *set1_id, int *set1_d_gain, int *set2, int *set2_id, int *set2_d_gain, int *gain_benefit, int *max_location, int MODULES, int set1_size, int set2_size, int k)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
	if(i < MODULES)
	{
		temp = set1[((gain_benefit[(*max_location)*3 + 0])*MODULES) + i];
		set1[((gain_benefit[(*max_location)*3 + 0])*MODULES) + i] = set1[((set1_size-1-k)*MODULES) + i];
		set1[((set1_size-1-k)*MODULES) + i] = temp;
	
		temp = set2[((gain_benefit[(*max_location)*3 + 1])*MODULES) + i];
		set2[((gain_benefit[(*max_location)*3 + 1])*MODULES) + i] = set2[((set2_size-1-k)*MODULES) + i];
		set2[((set2_size-1-k)*MODULES) + i] = temp;
	
		if(i == 0)
		{
				//------------------------------------------SWAP SET1 PARAMETERS
			temp = set1_id[gain_benefit[(*max_location)*3 + 0]];
			set1_id[gain_benefit[(*max_location)*3 + 0]] = set1_id[set1_size-1-k];
			set1_id[set1_size-1-k] = temp;
	
			temp = set1_d_gain[gain_benefit[(*max_location)*3 + 0]];
			set1_d_gain[gain_benefit[(*max_location)*3 + 0]] = set1_d_gain[set1_size-1-k];
			set1_d_gain[set1_size-1-k] = temp;
				//------------------------------------------SWAP SET2 PARAMETERS
			temp = set2_id[gain_benefit[(*max_location)*3 + 1]];
			set2_id[gain_benefit[(*max_location)*3 + 1]] = set2_id[set2_size-1-k];
			set2_id[set2_size-1-k] = temp;
	
			temp = set2_d_gain[gain_benefit[(*max_location)*3 + 1]];
			set2_d_gain[gain_benefit[(*max_location)*3 + 1]] = set2_d_gain[set2_size-1-k];
			set2_d_gain[set2_size-1-k] = temp;
		}
	}
}
//GPU KERNEL-----------------------------------------------------------------------GAIN UPDATE
__global__ void gain_update(int *set1, int *set2, int *set1_id, int *set2_id, int *set1_d_gain, int *set2_d_gain, int set1_size, int set2_size, int MODULES, int k)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < set1_size-k-1)
	{
		set1_d_gain[tid] = set1_d_gain[tid] + 2*(set1[((set1_size-1-k)*MODULES) + set1_id[tid]]) - 2*(set2[((set2_size-1-k)*MODULES) + set1_id[tid]]);
		if (tid < set2_size-k-1)
			set2_d_gain[tid] = set2_d_gain[tid] - 2*(set1[((set1_size-1-k)*MODULES) + set2_id[tid]]) + 2*(set2[((set2_size-1-k)*MODULES) + set2_id[tid]]);
	}
}
//GPU KERNEL-----------------------------------------------------------------------CALCULATE SET UPDATE LOCATION
__global__ void calc_set_update_location(int *iteration, int *set1_id, int *set2_id, int *set1t, int *set2t, int set1_size, int set2_size, int i)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(j < set1_size)
	{
		if(iteration[(i*3) + 0] == set1_id[j])
		{
			*set1t = j;
		}
	}
	
	if(j < set2_size)
	{
		if(iteration[(i*3) + 1] == set2_id[j])
		{
			*set2t = j;
		}
	}
}
//GPU KERNEL-----------------------------------------------------------------------SET UPDATE
__global__ void set_update(int *set1, int *set2, int *set1_id, int *set2_id, int *set1t, int *set2t, int MODULES)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
		
	if(j < MODULES)
	{
		temp = set1[((*set1t)*MODULES) + j];
		set1[((*set1t)*MODULES) + j] = set2[((*set2t)*MODULES) + j];
		set2[((*set2t)*MODULES) + j] = temp;
	}
	
	if(j == 0)
	{
		temp = set1_id[*set1t];
		set1_id[*set1t] = set2_id[*set2t];
		set2_id[*set2t] = temp;
	}
}
//-------------------------------------------------------------------------------MAIN
int main(int argc, char *argv[]) 
{
	int *set1_id;
	int *set2_id;
	int *set1_int_gain;
	int *set1_ext_gain;
	int *set2_int_gain;
	int *set2_ext_gain;
	int *set1_d_gain;
	int *set2_d_gain;
	int *set1;
	int *set2;
	int *gain_benefit;
	int *iteration;

	int *dev_set1_id;
	int *dev_set2_id;
	int *dev_set1_int_gain;
	int *dev_set1_ext_gain;
	int *dev_set2_int_gain;
	int *dev_set2_ext_gain;
	int *dev_set1_d_gain;
	int *dev_set2_d_gain;
	int *dev_set1;
	int *dev_set2;
	int *dev_gain_benefit;
	int *dev_max_g;
	int *dev_max_g_location;
	int *dev_maxt;
	int *dev_max_location;
	int *dev_iteration;
	int *dev_set1t;
	int *dev_set2t;
	
	FILE *fptr;
	int i,j,k,t, temp, maxt, max_location, max_new;
	int initial_cutset_size = 0;
	int final_cutset_size = 0;
	int len = 0;
	int value = 0;
	int location = 0;
	int PINS = 0;
	int NETS = 0;
	int MODULES = 0;
	int PADS = 0;
	int CELLS = 0;
	int check_mat[1000];
	int g_max[1000];
	int g_count = 0;
	int check_len = 0;
	int set1_size = 0;
	int set2_size = 0;
	char c;
	char s[20];
	char IN_FILE[100];
    //----------------------------------------------------------Command line arguement check
    if(argc < 2)
    {
    	printf("INPUT FORMAT: executable filename.net\n");
    	exit(0);
    }
	strcpy(IN_FILE, argv[1]);
    //----------------------------------------------------------Open IN_FILE 
    fptr = fopen(IN_FILE, "r"); 
    if (fptr == NULL) 
    { 
        printf("Cannot open file \n"); 
        exit(0); 
    } 
    //----------------------------------------------------------Read #PINS #NETS #MODULES #PADS
    fgets(s, sizeof(s), fptr);			//read first line
    c = getc(fptr); 					//read next character
    
    while(c != '\n')					//read number of PINS
    {
    	PINS = PINS*10 + ((int)c - 48);
    	c = getc(fptr);
    }
    c = getc(fptr);						//read next character
    while(c != '\n')					//read number of NETS
    {
    	NETS = NETS*10 + ((int)c - 48);
    	c = getc(fptr);
    }
    c = getc(fptr); 					//read next character
    while(c != '\n')					//read number of MODULES
    {
    	MODULES = MODULES*10 + ((int)c - 48);
    	c = getc(fptr);
    }
    c = getc(fptr); 					//read next character
    while(c != '\n')    				//read number of CELLS
    {
    	CELLS = CELLS*10 + ((int)c - 48);
    	c = getc(fptr);
    }
    PADS = MODULES - CELLS;				//calculate number of PADS
    
    printf("**************DATASET DETAILS**************\n");
    //printf("PINS: \t\t%d\n", PINS);
    //printf("NETS: \t\t%d\n", NETS);
    printf("MODULES: \t%d\n", MODULES);
    printf("PADS: \t\t%d\n", PADS);
    printf("CELLS: \t\t%d\n", CELLS);
    
    //---------------------------------------------------------- HOST MEMORY ALLOCATION
    //calculate the size of set1 and set2
    set2_size = MODULES/2;				
    set1_size = MODULES - set2_size;	//set1_size >= set2_size
    
    //allocate memory for det id
    set1_id = (int *)calloc(set1_size, sizeof(int)); 
    set2_id = (int *)calloc(set2_size, sizeof(int)); 
    
    //allocate memory for internal and external gain for both sets
    set1_int_gain = (int *)calloc(set1_size, sizeof(int)); 
    set1_ext_gain = (int *)calloc(set1_size, sizeof(int)); 
    set2_int_gain = (int *)calloc(set2_size, sizeof(int)); 
    set2_ext_gain = (int *)calloc(set2_size, sizeof(int)); 
    set1_d_gain = (int *)calloc(set1_size, sizeof(int)); 
    set2_d_gain = (int *)calloc(set2_size, sizeof(int)); 
    
    //allocate memory for set1
    set1 = (int *)calloc(set1_size*MODULES, sizeof(int)); 
    for (i=0; i<set1_size; i++)
    {
         set1_id[i] = i;
    }
    //allocate memory for set2
    set2 = (int *)calloc(set2_size*MODULES, sizeof(int)); 
    for (i=0; i<set2_size; i++) 
    {
         set2_id[i] = i + set1_size;
    }    
    //allocate memory for gain benefit, 3 columns
    //SET1_ID(array location ref) SET2_ID(array location ref) GAIN_BENEFIT
    t = set1_size*set2_size;
    gain_benefit = (int *)calloc(t*3, sizeof(int)); 
    
    //allocate memory to store result of all iterations
    //SET1_ID(0)  SET2_ID(1)  SWAP_BENEFIT(2)
    iteration = (int *)calloc(set2_size*3, sizeof(int));
         
    printf("********HOST MEMORY ALLOCATION COMPLETED********\n");
    //START-----------------------------------------------------PARSER
    while (fgets(s, sizeof(s), fptr))
    {
		len = strlen(s);
		
		if(argv[1][0] == 'i')			//IBM DATASET HAS AN EXTRA SPACE
		{
			len -= 1;
		}
		
		if(s[len-2] == '1')
		{
			if(check_len > 1)			//INNER CONNECTION FOR OLD NET
			{
				for(i=0; i<check_len; i++)
				{
					for(j=0; j<check_len; j++)
					{
						if(i != j)
						{
							if(check_mat[i] > (set1_size - 1))
								set2[((check_mat[i] - set1_size)*MODULES) + check_mat[j]] += 1;
							else
								set1[(check_mat[i])*MODULES + check_mat[j]] += 1;
						}
					}
				}
			}
			
			//NEW NET CONNECTION
			value = 0;
			check_len = 0;
			for(i=1; i<len-5; i++)
			{
				value = value*10 + ((int)s[i] - 48);
			}
		
			if(s[0] == 'a')
			{
				value += PADS;
			}
			else
			{
				value -= 1;
			}
			
			continue;
		}
		
		location = 0;
		for(i=1; i<len-3; i++)
		{
			location = location*10 + ((int)s[i] - 48);
		}
		if(s[0] == 'a')
		{
			location += PADS;
		}
		else
		{
			location -= 1;
		}	

		if(value > (set1_size - 1))
			set2[((value-set1_size)*MODULES) + location] += 1;
		else	
			set1[(value)*MODULES + location] += 1;
		if(location > (set1_size - 1))
			set2[((location-set1_size)*MODULES) + value] += 1;
		else	
			set1[(location)*MODULES + value] += 1;
			
		check_mat[check_len] = location;
		check_len++;
	}
	fclose(fptr);
    //END-------------------------------------------------------PARSER
    //START-----------------------------------------------------INITIAL GAIN CALCULATE
    hipMalloc((void**)&dev_set1_id, set1_size*sizeof(int));
    hipMalloc((void**)&dev_set2_id, set2_size*sizeof(int));
    hipMalloc((void**)&dev_set1_int_gain, set1_size*sizeof(int));
    hipMalloc((void**)&dev_set1_ext_gain, set1_size*sizeof(int));
    hipMalloc((void**)&dev_set2_int_gain, set2_size*sizeof(int));
    hipMalloc((void**)&dev_set2_ext_gain, set2_size*sizeof(int));
    hipMalloc((void**)&dev_set1_d_gain, set1_size*sizeof(int));
    hipMalloc((void**)&dev_set2_d_gain, set2_size*sizeof(int));
    hipMalloc((void**)&dev_set1, set1_size*MODULES*sizeof(int));
    hipMalloc((void**)&dev_set2, set2_size*MODULES*sizeof(int));
    hipMalloc((void**)&dev_gain_benefit, set1_size*set2_size*3*sizeof(int));
    
    temp = (int) sqrt(set1_size*set2_size);
    hipMalloc((void**)&dev_max_g, temp*sizeof(int));
    hipMalloc((void**)&dev_max_g_location, temp*sizeof(int));
    hipMalloc((void**)&dev_maxt, sizeof(int));
    hipMalloc((void**)&dev_max_location, sizeof(int));
    hipMalloc((void**)&dev_iteration, set2_size*3*sizeof(int));
    hipMalloc((void**)&dev_set1t, sizeof(int));
    hipMalloc((void**)&dev_set2t, sizeof(int));
    printf("*******DEVICE MEMORY ALLOCATION COMPLETED*******\n");
    hipMemcpy(dev_set1_id, set1_id, set1_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set2_id, set2_id, set2_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set1_int_gain, set1_int_gain, set1_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set1_ext_gain, set1_ext_gain, set1_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set2_int_gain, set2_int_gain, set2_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set2_ext_gain, set2_ext_gain, set2_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set1_d_gain, set1_d_gain, set1_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set2_d_gain, set2_d_gain, set2_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set1, set1, set1_size*MODULES*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_set2, set2, set2_size*MODULES*sizeof(int), hipMemcpyHostToDevice);
    
    //START-----------------------------------------------------PRINT SET ID
    /*
    printf("-SET1--SET2-\n");
    for(i=0; i<set1_size; i++)
    {
    	if(i < set2_size)
    	{
    		printf("%d\t%d\n", set1_id[i], set2_id[i]);
    	}
    	else
    	{
    		printf("%d\n", set1_id[i]);
    	}
    }
    */
    //END-------------------------------------------------------PRINT SET ID
    //START-----------------------------------------------------PRINT SET
    /*
    printf("----------SET1----------\n");
    for(i=0; i<set1_size; i++)
    {
    	for(j=0; j<MODULES; j++)
    	{
    		printf("%d  ", set1[(i*MODULES)+j]);
    	}
    	printf("\n");
    }
    printf("----------SET2----------\n");
    for(i=0; i<set2_size; i++)
    {
    	for(j=0; j<MODULES; j++)
    	{
    		printf("%d  ", set2[(i*MODULES)+j]);
    	}
    	printf("\n");
    } 
    */
    //END-------------------------------------------------------PRINT SET
    int block_num, thread_num;
    //START-----------------------------------------------------INITIAL GAIN CALCULATE   
    block_num = ceil((double)set1_size/MAX_TH);

    calc_init_gain<<<block_num,MAX_TH>>>(dev_set1, dev_set2, dev_set1_int_gain, dev_set1_ext_gain, dev_set2_int_gain, dev_set2_ext_gain, dev_set1_d_gain, dev_set2_d_gain, dev_set1_id, dev_set2_id, set1_size, set2_size, MODULES);
    
    hipMemcpy(set2_ext_gain, dev_set2_ext_gain, set2_size*sizeof(int), hipMemcpyDeviceToHost);

	for(i=0; i<set2_size; i++)
    {
    	initial_cutset_size += set2_ext_gain[i];
    }
    //END-------------------------------------------------------INITIAL GAIN CALCULATE   
   	while(1)
    {
    	printf("------------------------------------RUN %d\n", g_count+1);
		for(k=0; k<set2_size; k++)
		{
			//START-----------------------------------------------------CALCULATE GAIN BENEFIT		
			dim3 blocksize(32, 32, 1);
			dim3 gridsize (ceil((double)(set2_size-k)/32), ceil((double)(set1_size-k)/32), 1);
	  		  		
			cal_gain_benefit<<<gridsize,blocksize>>>(dev_set1, dev_set2_id, dev_set1_d_gain, dev_set2_d_gain, dev_gain_benefit, set1_size, set2_size, k, MODULES);
			//END-------------------------------------------------------CALCULATE GAIN BENEFIT
			//START-----------------------------------------------------CALCULATE MAXIMUM GAIN SWAP
			t = (set1_size-k)*(set2_size-k);
			thread_num = (int) sqrt(t);
			block_num = ceil((double)thread_num/MAX_TH);
			temp = ceil((double)t/thread_num);
			
			//printf("thread_num: %d\n", thread_num);
			//printf("block_num: %d\n", block_num);
			//printf("temp: %d\n", temp);
		
			cal_max_g_t<<<block_num,MAX_TH>>>(dev_gain_benefit, dev_max_g, dev_max_g_location, t, temp, thread_num);
			cal_max_g <<<1,1>>>(dev_max_g, dev_max_g_location, dev_maxt, dev_max_location, thread_num);
			update_iteration <<<1,1>>>(dev_iteration, dev_set1_id, dev_set2_id, dev_gain_benefit, dev_max_location, dev_maxt, k);	
			//END-------------------------------------------------------CALCULATE MAXIMUM GAIN SWAP	
			//START-----------------------------------------------------SWAP
			block_num = ceil((double)MODULES/MAX_TH);
			gain_swap <<<block_num,MAX_TH>>>(dev_set1, dev_set1_id, dev_set1_d_gain, dev_set2, dev_set2_id, dev_set2_d_gain, dev_gain_benefit, dev_max_location, MODULES, set1_size, set2_size, k);	
			//END-------------------------------------------------------SWAP
			//START-----------------------------------------------------UPDATE GAIN (ONLY D)		
			if(k+1 < set2_size)
			{
				block_num = ceil((double)(set1_size-k-1)/MAX_TH);
				gain_update<<<block_num,MAX_TH>>>(dev_set1, dev_set2, dev_set1_id, dev_set2_id, dev_set1_d_gain, dev_set2_d_gain, set1_size, set2_size, MODULES, k);
			}
			//END-------------------------------------------------------UPDATE GAIN (ONLY D)
		}	
		//----------------------------------------------------------CALCULATE MAX CUMULATIVE GAIN
		hipMemcpy(iteration, dev_iteration, set2_size*3*sizeof(int), hipMemcpyDeviceToHost);
		maxt = 0;
		max_new = 0;
		max_location = 0;
		for(i=0; i<set2_size; i++)
		{
			max_new += iteration[(i*3) + 2];
			if(max_new > maxt)
			{
				maxt = max_new;
				max_location = i;
			}
		}
		//----------------------------------------------------------SWAP TO GET UPDATED SETS
		if(maxt > 0)
		{
			g_max[g_count] = maxt;
			g_count++;
			for(i=0; i<=max_location; i++)
			{
				block_num = ceil((double)set1_size/MAX_TH);
				calc_set_update_location<<<block_num,MAX_TH>>>(dev_iteration, dev_set1_id, dev_set2_id, dev_set1t, dev_set2t, set1_size, set2_size, i);
								
				block_num = ceil((double)MODULES/MAX_TH);
				set_update<<<block_num,MAX_TH>>>(dev_set1, dev_set2, dev_set1_id, dev_set2_id, dev_set1t, dev_set2t, MODULES);
			}
			//----------------------------------------------------------INITIAL GAIN FOR NEXT RUN
			block_num = ceil((double)set1_size/MAX_TH);
			calc_init_gain<<<block_num,MAX_TH>>>(dev_set1, dev_set2, dev_set1_int_gain, dev_set1_ext_gain, dev_set2_int_gain, dev_set2_ext_gain, dev_set1_d_gain, dev_set2_d_gain, dev_set1_id, dev_set2_id, set1_size, set2_size, MODULES);
				
		}
		else
		{
			/*
			cudaMemcpy(set1_id, dev_set1_id, set1_size*sizeof(int), cudaMemcpyDeviceToHost);
			cudaMemcpy(set2_id, dev_set2_id, set2_size*sizeof(int), cudaMemcpyDeviceToHost);
			cudaMemcpy(set1, dev_set1, set1_size*MODULES*sizeof(int), cudaMemcpyDeviceToHost);
			cudaMemcpy(set2, dev_set2, set2_size*MODULES*sizeof(int), cudaMemcpyDeviceToHost);
			*/
			break;
		}
	
	}
	//----------------------------------------------------------CALCULATE TOTAL GAIN OF ALL RUNS
	maxt = 0;
	for(i=0; i<g_count; i++)
	{
		maxt += g_max[i];
	}
	//----------------------------------------------------------CALCULATE FINAL CUT-SET SIZE
	final_cutset_size = initial_cutset_size - maxt;
	
	printf("****************FINAL RESULT***************\n");
	printf("Max Cumulative Gain: \t\t%d\n", maxt);
	//printf("Max Cumulative Gain Iteration: \t%d\n", max_location+1);
	printf("Initial Cutset Size: \t\t%d\n", initial_cutset_size);
    printf("Final Cutset Size: \t\t%d\n", final_cutset_size);
    printf("Number of Global Run: \t\t%d\n", g_count+1);
    printf("*******************************************\n");
    //START-----------------------------------------------------PRINT SET ID
    /*
    printf("------------\n");
    printf("-SET1--SET2-\n");
    for(i=0; i<set1_size; i++)
    {
    	if(i < set2_size)
    	{
    		printf("%d\t%d\n", set1_id[i], set2_id[i]);
    	}
    	else
    	{
    		printf("%d\n", set1_id[i]);
    	}
    }
    */
    //END-------------------------------------------------------PRINT SET ID
    //START-----------------------------------------------------PRINT SET
    /*
    printf("----------SET1----------\n");
    for(i=0; i<set1_size; i++)
    {
    	for(j=0; j<MODULES; j++)
    	{
    		printf("%d  ", set1[(i*MODULES)+j]);
    	}
    	printf("\n");
    }
    printf("----------SET2----------\n");
    for(i=0; i<set2_size; i++)
    {
    	for(j=0; j<MODULES; j++)
    	{
    		printf("%d  ", set2[(i*MODULES)+j]);
    	}
    	printf("\n");
    }   
    */
    //END-------------------------------------------------------PRINT SET
    //START-----------------------------------------------------FREE MEMORY
    free(set1);
    free(set2);
    free(set1_id);
    free(set2_id);
    free(set1_int_gain);
    free(set1_ext_gain);
    free(set2_int_gain);
    free(set2_ext_gain);
    free(set1_d_gain);
    free(set2_d_gain);
    free(gain_benefit);
    free(iteration);
    
    hipFree(dev_set1_id);
	hipFree(dev_set2_id);
	hipFree(dev_set1_int_gain);
	hipFree(dev_set1_ext_gain);
	hipFree(dev_set2_int_gain);
	hipFree(dev_set2_ext_gain);
	hipFree(dev_set1_d_gain);
	hipFree(dev_set2_d_gain);
	hipFree(dev_set1);
	hipFree(dev_set2);
	hipFree(dev_gain_benefit);
	hipFree(dev_max_g);
	hipFree(dev_max_g_location);
	hipFree(dev_maxt);
	hipFree(dev_max_location);
	hipFree(dev_iteration);
	hipFree(dev_set1t);
	hipFree(dev_set2t);
  
return 0; 
}
